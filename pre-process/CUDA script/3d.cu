#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

 #include "forward.h"
 #include "auxiliary.h"
 #include <hip/hip_cooperative_groups.h>
 #include <cooperative_groups/reduce.h>
 namespace cg = cooperative_groups;
 
 __device__ __constant__ float pi = 3.14159265358979323846f;

 
 
 // Forward version of 2D covariance matrix computation
 __device__ float3 computeCov2D(const float3& mean, float focal_x, float focal_y, float tan_fovx, float tan_fovy, const float* cov3D, const float* viewmatrix)
 {
	 // The following models the steps outlined by equations 29
	 // and 31 in "EWA Splatting" (Zwicker et al., 2002). 
	 // Additionally considers aspect / scaling of viewport.
	 // Transposes used to account for row-/column-major conventions.
	 float3 t = transformPoint4x3(mean, viewmatrix);
 
	 const float limx = 1.3f * tan_fovx;
	 const float limy = 1.3f * tan_fovy;
	 const float txtz = t.x / t.z;
	 const float tytz = t.y / t.z;
	 t.x = min(limx, max(-limx, txtz)) * t.z;
	 t.y = min(limy, max(-limy, tytz)) * t.z;
 
	 glm::mat3 J = glm::mat3(
		 focal_x / t.z, 0.0f, -(focal_x * t.x) / (t.z * t.z),
		 0.0f, focal_y / t.z, -(focal_y * t.y) / (t.z * t.z),
		 0, 0, 0);
 
	 glm::mat3 W = glm::mat3(
		 viewmatrix[0], viewmatrix[4], viewmatrix[8],
		 viewmatrix[1], viewmatrix[5], viewmatrix[9],
		 viewmatrix[2], viewmatrix[6], viewmatrix[10]);
 
	 glm::mat3 T = W * J;
 
	 glm::mat3 Vrk = glm::mat3(
		 cov3D[0], cov3D[1], cov3D[2],
		 cov3D[1], cov3D[3], cov3D[4],
		 cov3D[2], cov3D[4], cov3D[5]);
 
	 glm::mat3 cov = glm::transpose(T) * glm::transpose(Vrk) * T;
 
	 // Apply low-pass filter: every Gaussian should be at least
	 // one pixel wide/high. Discard 3rd row and column.
	 cov[0][0] += 0.3f;
	 cov[1][1] += 0.3f;
	 return { float(cov[0][0]), float(cov[0][1]), float(cov[1][1]) };
 }
 
 // Forward method for converting scale and rotation properties of each
 // Gaussian to a 3D covariance matrix in world space. Also takes care
 // of quaternion normalization.
 __device__ void computeCov3D(const glm::vec3 scale, float mod, const glm::vec4 rot, float* cov3D)
 {
	 // Create scaling matrix
	 glm::mat3 S = glm::mat3(1.0f);
	 S[0][0] = mod * scale.x;
	 S[1][1] = mod * scale.y;
	 S[2][2] = mod * scale.z;
 
	 // Normalize quaternion to get valid rotation
	 glm::vec4 q = rot;// / glm::length(rot);
	 float r = q.x;
	 float x = q.y;
	 float y = q.z;
	 float z = q.w;
 
	 // Compute rotation matrix from quaternion
	 glm::mat3 R = glm::mat3(
		 1.f - 2.f * (y * y + z * z), 2.f * (x * y - r * z), 2.f * (x * z + r * y),
		 2.f * (x * y + r * z), 1.f - 2.f * (x * x + z * z), 2.f * (y * z - r * x),
		 2.f * (x * z - r * y), 2.f * (y * z + r * x), 1.f - 2.f * (x * x + y * y)
	 );
 
	 glm::mat3 M = S * R;
 
	 // Compute 3D world covariance matrix Sigma
	 glm::mat3 Sigma = glm::transpose(M) * M;
 
	 // Covariance is symmetric, only store upper right
	 cov3D[0] = Sigma[0][0];
	 cov3D[1] = Sigma[0][1];
	 cov3D[2] = Sigma[0][2];
	 cov3D[3] = Sigma[1][1];
	 cov3D[4] = Sigma[1][2];
	 cov3D[5] = Sigma[2][2];
 }
 
 // Perform initial steps for each Gaussian prior to rasterization.
 template<int C>
 __global__ void preprocessCUDA(int P, int D, int M,
	 const float* orig_points,
	 const glm::vec3* scales,
	 const float scale_modifier,
	 const glm::vec4* rotations,
	 const float* opacities,
	 const float* shs,
	 bool* clamped,
	 const float* cov3D_precomp,
	 const float* colors_precomp,
	 const float* viewmatrix,
	 const float* projmatrix,
	 const glm::vec3* cam_pos,
	 const int W, int H,
	 const float tan_fovx, float tan_fovy,
	 const float focal_x, float focal_y,
	 int* radii,
	 float2* points_xy_image,
	 float* depths,
	 float* cov3Ds,
	 float* rgb,
	 float4* conic_opacity,
	 const dim3 grid,
	 uint32_t* tiles_touched,
	 bool prefiltered)
	 auto idx = cg::this_grid().thread_rank();
	 if (idx >= P)
		 return;
 
	 // Initialize radius and touched tiles to 0. If this isn't changed,
	 // this Gaussian will not be processed further.
	 radii[idx] = 0;
	 tiles_touched[idx] = 0;
 
	 // Perform near culling, quit if outside.
	 float3 p_view;
	 if (!in_frustum(idx, orig_points, viewmatrix, projmatrix, prefiltered, p_view))
		 return;
 
	 // Transform point by projecting
	 float3 p_orig = { orig_points[3 * idx], orig_points[3 * idx + 1], orig_points[3 * idx + 2] };
 
	 // If 3D covariance matrix is precomputed, use it, otherwise compute
	 // from scaling and rotation parameters. 
	 const float* cov3D;
	 if (cov3D_precomp != nullptr)
	 {
		 cov3D = cov3D_precomp + idx * 6;
	 }
	 else
	 {
		 computeCov3D(scales[idx], scale_modifier, rotations[idx], cov3Ds + idx * 6);
		 cov3D = cov3Ds + idx * 6;
	 }
 
	 // Compute 2D screen-space covariance matrix
	 float3 cov = computeCov2D(p_orig, focal_x, focal_y, tan_fovx, tan_fovy, cov3D, viewmatrix);
 
	 // Invert covariance (EWA algorithm)
	 float det = (cov.x * cov.z - cov.y * cov.y);
	 if (det == 0.0f)
		 return;
	 float det_inv = 1.f / det;
	 float3 conic = { cov.z * det_inv, -cov.y * det_inv, cov.x * det_inv };
 